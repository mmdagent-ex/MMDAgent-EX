#include "hip/hip_runtime.h"
/*

 * Copyright (c) 1991-2020 Kawahara Lab., Kyoto University
 * Copyright (c) 2000-2005 Shikano Lab., Nara Institute of Science and Technology
 * Copyright (c) 2005-2020 Julius project team, Nagoya Institute of Technology
 * All rights reserved
 */

#include <sent/stddefs.h>
#include <sent/htk_hmm.h>
#include <sent/htk_param.h>
#include <sent/hmm.h>
#include <sent/hmm_calc.h>

#ifdef __NVCC__

#include <hip/hip_runtime.h>

// from common.h
#include <sys/time.h>

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

/* define this to test disabling expsum computation at softmax */
#undef NO_SUM_COMPUTATION

/***********************************************************************/
/*
 * global mode: not use shared memory, block size is BLOCK_SIZE
 *
 * shared mode: use [BLOCK_SIZE_X][BLOCK_SIZE_Y] threads
 *
 */

#define BLOCK_SIZE 128    /// Block size in global mode
#define BLOCK_SIZE_X 16   /// Block size X in shared mode
#define BLOCK_SIZE_Y 8    /// Block size Y in shared mode

typedef struct {
  int stride;
  float *elements_src;
  float *elements_dst;
} Matrix;

/***********************************************************************/
/* should be same value as calc_dnn.c */
#define LOGISTIC_TABLE_FACTOR 20000
#define LOGISTIC_TABLE_MAX (16 * LOGISTIC_TABLE_FACTOR)
#define LOGISTIC_MIN 0.000334
#define LOGISTIC_MAX 0.999666

static float *d_logistic;  /* GPU entry point of logistic value table */

/* build logistic function value table on GPU */
void cuda_copy_logistic_table(float *table, int len)
{
  // copy logistic_table to GPU
  CHECK(hipMalloc((void **)&d_logistic, sizeof(float) * len));
  CHECK(hipMemcpy(d_logistic, table, sizeof(float) * len, hipMemcpyHostToDevice));
}

/***********************************************************************/
// allocate GPU memory per DNN layer
void cuda_layer_load(DNNLayer *l)
{
  CHECK(hipMalloc((void **)&l->dw, sizeof(float) * l->out * l->in));
  CHECK(hipMemcpy(l->dw, l->w, sizeof(float) * l->out * l->in, hipMemcpyHostToDevice));
  CHECK(hipMalloc((void **)&l->db, sizeof(float) * l->out));
  CHECK(hipMemcpy(l->db, l->b, sizeof(float) * l->out, hipMemcpyHostToDevice));
}

// free GPU memory per DNN layer
void cuda_layer_free(DNNLayer *l)
{
  if (l->dw != NULL) CHECK(hipFree(l->dw));
  if (l->db != NULL) CHECK(hipFree(l->db));
}

// clear GPU part of DNN structure
void cuda_dnn_clear(DNNData *dnn)
{
  int i;

  if (dnn->ddst) {
    for (i = 0; i < dnn->hnum; i++) {
      if (dnn->ddst[i]) {
	CHECK(hipFree(dnn->ddst[i]));
      }
    }
    free(dnn->ddst);
  }
  if (dnn->dout) CHECK(hipFree(dnn->dout));
  if (dnn->dinvec) CHECK(hipFree(dnn->dinvec));
}

// set up
void cuda_dnn_setup(DNNData *dnn)
{
  int i;

  dnn->ddst = (float **)mymalloc(sizeof(float *) * dnn->hnum);
  for (i = 0; i < dnn->hnum; i++) {
    CHECK(hipMalloc((void **)&(dnn->ddst[i]), sizeof(float) * dnn->hiddennodenum));
  }
  CHECK(hipMalloc((void **)&dnn->dout, sizeof(float) * dnn->outputnodenum));
  CHECK(hipMalloc((void **)&dnn->dinvec, sizeof(float) * dnn->inputnodenum));

  if (dnn->use_cuda_shared == FALSE) {
    if (dnn->blocksize1 == 0) {
      dnn->blocksize1 = BLOCK_SIZE;
    }
  } else {
    if (dnn->blocksize1 == 0) {
      dnn->blocksize1 = BLOCK_SIZE_X;
    }
    if (dnn->blocksize2 == 0) {
      dnn->blocksize2 = BLOCK_SIZE_Y;
    }
  }
}

/***********************************************************************/
/* sigmoid computation on GPU*/

__global__ void _cuda_sigmoid(float *dst, float *logistic, int out)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < out) {
    if (dst[idx] <= -8.0f) {
      dst[idx] = LOGISTIC_MIN;
    } else if (dst[idx] >=  8.0f) {
      dst[idx] = LOGISTIC_MAX;
    } else {
      dst[idx] = logistic[(int)((dst[idx] + 8.0f) * LOGISTIC_TABLE_FACTOR + 0.5)];
    }
  }
}

/* calc DNN on GPU (global) */
__global__ void _cuda_calc_dnn(float *src, float *dst, float *w, float *b, int in, int out)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < out) {
    float x = 0.0f;
    for (int k = 0; k < in; ++k) {
      x += src[k] * w[idx * in + k];
    }
    dst[idx] = x + b[idx];
  }
}

/* global version */
static void local_calc_outprob_global(HMMWork *wrk)
{
  DNNData *dnn = wrk->OP_dnn;
  DNNLayer *h;

  // define <grid, block> for layer computation
  dim3 block(dnn->blocksize1);
  dim3 grid((dnn->hiddennodenum + block.x - 1) / block.x);

  // define <grid2, block2> for output layer computation
  dim3 block2(dnn->blocksize1);
  dim3 grid2((dnn->outputnodenum + block2.x - 1) / block2.x);

  // transfer input vectors to GPU
  CHECK(hipMemcpy(dnn->dinvec,
                   &(wrk->OP_param->parvec[wrk->OP_time][0]),
                   sizeof(float) * dnn->inputnodenum,
                   hipMemcpyHostToDevice));

  // do calculation on GPU
  h = &(dnn->h[0]);
  _cuda_calc_dnn<<<grid, block>>>(dnn->dinvec, dnn->ddst[0], h->dw, h->db, h->in, h->out);
  _cuda_sigmoid<<<grid, block>>>(dnn->ddst[0], d_logistic, h->out);

  for (int hidx = 1; hidx < dnn->hnum; hidx++) {
    h = &(dnn->h[hidx]);
    _cuda_calc_dnn<<<grid, block>>>(dnn->ddst[hidx-1], dnn->ddst[hidx], h->dw, h->db, h->in, h->out);
    _cuda_sigmoid<<<grid, block>>>(dnn->ddst[hidx], d_logistic, h->out);
  }

  _cuda_calc_dnn<<<grid2, block2>>>(dnn->ddst[dnn->hnum-1], dnn->dout, dnn->o.dw, dnn->o.db, dnn->o.in, dnn->o.out);

  /* transfer result from GPU to cpu */
  CHECK(hipMemcpy(wrk->last_cache, dnn->dout, sizeof(float) * dnn->outputnodenum, hipMemcpyDeviceToHost));

}

/***********************************************************************/

/* return first point of partial matrix that corresponds to the block */
__device__ Matrix GetSubMatrix(float *A, int row, int col, int in)
{
  Matrix Asub;

  Asub.elements_src = &A[in * BLOCK_SIZE_Y * row + BLOCK_SIZE_X * col];
  Asub.elements_dst = &A[in * BLOCK_SIZE_X * row + BLOCK_SIZE_Y * col];

  return Asub;
}

/* compute a block */
__global__ void _cuda_calc_dnn_shared(float *src, float *dst, float *w, float *b, int in, int out)
{
  int brow = blockIdx.y; /* should be always 0 since this is 1-dim. grid */
  int bcol = blockIdx.x; /* block ID */

  int trow = threadIdx.y; /* 0 ... BLOCK_SIZE_Y-1 */
  int tcol = threadIdx.x; /* 0 ... BKOCK_SIZE_X-1 */

  /* check if this is my part */
  if (bcol * BLOCK_SIZE_Y + trow < out) {

    /* take partial matrix */
    Matrix dst_sub = GetSubMatrix(dst, brow, bcol, in);

    /* MA loop */
    float x = 0.0f;
    for (int l = 0; l * BLOCK_SIZE_X + tcol < in; ++l) {
      /* get partial matrix of src and W */
      Matrix src_sub = GetSubMatrix(src, brow, l, in);
      Matrix W_sub = GetSubMatrix(w, bcol, l, in);

      /* put them to shared memory */
      __shared__ float srcs[BLOCK_SIZE_X];
      __shared__ float Ws[BLOCK_SIZE_Y][BLOCK_SIZE_X];
      srcs[tcol] = src_sub.elements_src[tcol];
      Ws[trow][tcol] = W_sub.elements_src[trow*in+tcol];
      __syncthreads();

      /* do matrix computation */
      for (int k = 0; k < BLOCK_SIZE_X; ++k) {
	x += srcs[k] * Ws[trow][k];
      }
      __syncthreads();
    }
    /* add bias vector */
    dst_sub.elements_dst[trow] = x + b[bcol * BLOCK_SIZE_Y + trow];
  }
}

/* shared version */
static void local_calc_outprob_shared(HMMWork *wrk)
{
  DNNData *dnn = wrk->OP_dnn;
  DNNLayer *h;

  // define <grid, block> for layer computation
  dim3 block(BLOCK_SIZE_X, BLOCK_SIZE_Y);
  dim3 grid((dnn->hiddennodenum + block.y - 1) / block.y, 1);

  // define <grid2, block2> for output layer computation
  dim3 block2(BLOCK_SIZE_X, BLOCK_SIZE_Y);
  dim3 grid2((dnn->outputnodenum + block2.y - 1) / block2.y, 1);

  // transfer input vectors to GPU
  CHECK(hipMemcpy(dnn->dinvec,
                   &(wrk->OP_param->parvec[wrk->OP_time][0]),
                   sizeof(float)*dnn->inputnodenum,
                   hipMemcpyHostToDevice));

  // do calculation on GPU
  h = &(dnn->h[0]);
  _cuda_calc_dnn_shared<<<grid, block>>>(dnn->dinvec, dnn->ddst[0], h->dw, h->db, h->in, h->out);
  _cuda_sigmoid<<<grid, block>>>(dnn->ddst[0], d_logistic, h->out);

  for (int hidx = 1; hidx < dnn->hnum; hidx++) {
    h = &(dnn->h[hidx]);
    _cuda_calc_dnn_shared<<<grid, block>>>(dnn->ddst[hidx-1], dnn->ddst[hidx], h->dw, h->db, h->in, h->out);
    _cuda_sigmoid<<<grid, block>>>(dnn->ddst[hidx], d_logistic, h->out);
  }

  _cuda_calc_dnn_shared<<<grid2, block2>>>(dnn->ddst[dnn->hnum-1], dnn->dout, dnn->o.dw, dnn->o.db, dnn->o.in, dnn->o.out);

  /* transfer result from GPU to cpu */
  CHECK(hipMemcpy(wrk->last_cache, dnn->dout, sizeof(float) * dnn->outputnodenum, hipMemcpyDeviceToHost));

}

/************************************************************************/
void cuda_calc_outprob(HMMWork *wrk)
{
  DNNData *dnn = wrk->OP_dnn;

  if (dnn->use_cuda_shared == true) {
    local_calc_outprob_shared(wrk);
  } else {
    local_calc_outprob_global(wrk);
  }

  /* do softmax */
  /* INV_LOG_TEN * (x - addlogarray(x)) - log10(state_prior)) */
#ifdef NO_SUM_COMPUTATION
  /* not compute sum */
  for (int i = 0; i < wrk->statenum; i++) {
    wrk->last_cache[i] = INV_LOG_TEN * wrk->last_cache[i] - dnn->state_prior[i];
  }
#else
  /* compute sum */
  {
    int i;
    float logprob = addlog_array(wrk->last_cache, wrk->statenum);
    for (i = 0; i < wrk->statenum; i++) {
      wrk->last_cache[i] = INV_LOG_TEN * (wrk->last_cache[i] - logprob) - dnn->state_prior[i];
    }
  }
#endif /* NO_SUM_COMPUTATION */
}

#endif /* __NVCC__ */
